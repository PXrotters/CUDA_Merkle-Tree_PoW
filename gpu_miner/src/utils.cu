#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include "utils.h"
#include <string.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// CUDA sprintf alternative for nonce finding. Converts integer to its string representation. Returns string's length.
__device__ int intToString(uint64_t num, char* out) {
    if (num == 0) {
        out[0] = '0';
        out[1] = '\0';
        return 2;
    }

    int i = 0;
    while (num != 0) {
        int digit = num % 10;
        num /= 10;
        out[i++] = '0' + digit;
    }

    // Reverse the string
    for (int j = 0; j < i / 2; j++) {
        char temp = out[j];
        out[j] = out[i - j - 1];
        out[i - j - 1] = temp;
    }
    out[i] = '\0';
    return i;
}

// CUDA strlen implementation.
__host__ __device__ size_t d_strlen(const char *str) {
    size_t len = 0;
    while (str[len] != '\0') {
        len++;
    }
    return len;
}

// CUDA strcpy implementation.
__device__ void d_strcpy(char *dest, const char *src){
    int i = 0;
    while ((dest[i] = src[i]) != '\0') {
        i++;
    }
}

// CUDA strcat implementation.
__device__ void d_strcat(char *dest, const char *src){
    while (*dest != '\0') {
        dest++;
    }
    while (*src != '\0') {
        *dest = *src;
        dest++;
        src++;
    }
    *dest = '\0';
}

// Compute SHA256 and convert to hex
__host__ __device__ void apply_sha256(const BYTE *input, BYTE *output) {
    size_t input_length = d_strlen((const char *)input);
    SHA256_CTX ctx;
    BYTE buf[SHA256_BLOCK_SIZE];
    const char hex_chars[] = "0123456789abcdef";

    sha256_init(&ctx);
    sha256_update(&ctx, input, input_length);
    sha256_final(&ctx, buf);

    for (size_t i = 0; i < SHA256_BLOCK_SIZE; i++) {
        output[i * 2]     = hex_chars[(buf[i] >> 4) & 0x0F];  // High nibble
        output[i * 2 + 1] = hex_chars[buf[i] & 0x0F];         // Low nibble
    }
    output[SHA256_BLOCK_SIZE * 2] = '\0'; // Null-terminate
}

// Compare two hashes
__host__ __device__ int compare_hashes(BYTE* hash1, BYTE* hash2) {
    for (int i = 0; i < SHA256_HASH_SIZE; i++) {
        if (hash1[i] < hash2[i]) {
            return -1; // hash1 is lower
        } else if (hash1[i] > hash2[i]) {
            return 1; // hash2 is lower
        }
    }
    return 0; // hashes are equal
}

// Kernel pentru a calcula SHA-256 pentru fiecare tranzactie
__global__ void kernel_hash_transactions(BYTE *transactions, BYTE *hashes, int transaction_size, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) {
        return;
    }
    apply_sha256(transactions + idx * transaction_size, hashes + idx * SHA256_HASH_SIZE);
}

// Functie pentru a construi radacina Merkle
void construct_merkle_root(int transaction_size, BYTE *transactions, int max_transactions_in_a_block, int n, BYTE merkle_root[SHA256_HASH_SIZE]) {
    BYTE *device_transactions, *device_hashes;

    // Alocam memorie pentru hash-uri pe gazda
    BYTE *hashes = (BYTE *)malloc(max_transactions_in_a_block * SHA256_HASH_SIZE);

    // Alocam memorie pentru tranzactii si hash-uri pe GPU
    hipMalloc(&device_transactions, n * transaction_size);
    hipMalloc(&device_hashes, n * SHA256_HASH_SIZE);
    
    // Copiem tranzactiile pe GPU
    hipMemcpy(device_transactions, transactions, n * transaction_size, hipMemcpyHostToDevice);

    // 1) Calculam hash-urile pentru fiecare tranzactie
    int threads_per_block = 256;
    int blocks = (n + threads_per_block - 1) / threads_per_block;
    kernel_hash_transactions<<<blocks, threads_per_block>>>(device_transactions, device_hashes, transaction_size, n);
    hipDeviceSynchronize();

    // Copiem inapoi hash-urile de pe GPU pe CPU si eliberam memoria GPU
    hipMemcpy(hashes, device_hashes, n * SHA256_HASH_SIZE, hipMemcpyDeviceToHost);
    hipFree(device_transactions);
    hipFree(device_hashes);

    // 2) Reducem hash-urile pentru a obtine radacina Merkle
    while (n > 1) {
        int nr = 0;
        // Pentru fiecare pereche de hash-uri, le combinam si calculam hash-ul rezultat (sau duplicam ultimul hash daca n e impar)
        for (int i = 0; i < n; i += 2) {
            BYTE combined[SHA256_HASH_SIZE * 2];  // Buffer pentru a combina doua hash-uri
            if (i + 1 < n) {
                memcpy(combined, &hashes[i * SHA256_HASH_SIZE], SHA256_HASH_SIZE);  // Copiem primul hash
                memcpy(combined + SHA256_HASH_SIZE, &hashes[(i + 1) * SHA256_HASH_SIZE], SHA256_HASH_SIZE);  // Copiem al doilea hash
            } else {
                memcpy(combined, &hashes[i * SHA256_HASH_SIZE], SHA256_HASH_SIZE);  // Copiem ultimul hash
                memcpy(combined + SHA256_HASH_SIZE, &hashes[i * SHA256_HASH_SIZE], SHA256_HASH_SIZE);  // Duplicam ultimul hash
            }
            // Aplicam SHA256 pe hash-urile combinate
            apply_sha256(combined, &hashes[nr * SHA256_HASH_SIZE]);
            nr++;
        }
        n = nr;  // Actualizam numarul de hash-uri
    }

    // 3) Copiem radacina Merkle in merkle_root
    memcpy(merkle_root, hashes, SHA256_HASH_SIZE);
    free(hashes);
}

// Kernel pentru a gasi nonce-ul valid
__global__ void find_valid_nonce(BYTE *difficulty, BYTE *block_content, size_t content_length, uint32_t max_nonce, int *found_flag, uint32_t *found_nonce, BYTE *resultingHash) {
    uint32_t nonce = blockIdx.x * blockDim.x + threadIdx.x;

    if (nonce > max_nonce || *found_flag) {
        return;
    }

    // Cream un bloc local pentru a concatena continutul blocului si nonce-ul
    char local_block[BLOCK_SIZE];
    char nonce_str[NONCE_SIZE];
    BYTE computed_hash[SHA256_HASH_SIZE];

    // Copiem continutul blocului in local_block
    for (int i = 0; i < content_length; ++i) {
        local_block[i] = block_content[i];
    }
    local_block[content_length] = '\0';

    // Convertim nonce-ul in string si il adaugam la local_block
    int nonce_len = intToString(nonce, nonce_str);
    for (int i = 0; i < nonce_len; ++i) {
        local_block[content_length + i] = nonce_str[i];
    }
    local_block[content_length + nonce_len] = '\0';

    // Aplicam SHA256 pe local_block
    apply_sha256((BYTE *)local_block, computed_hash);

    // Verificam daca hash-ul este mai mic decat dificultatea
    if (compare_hashes(computed_hash, difficulty) <= 0) {
        // Daca e primul nonce gasit, il salvam si setam flag-ul
        int old_value = atomicExch(found_flag, 1);
        if (old_value == 0) {
            *found_nonce = nonce;  // Retinem valoarea nonce-ului gasit
            // Copiem hash-ul gasit in resultingHash
            for (int i = 0; i < SHA256_HASH_SIZE; ++i) {
                resultingHash[i] = computed_hash[i];
            }
        }
    }
}

// Functie pentru a gasi nonce-ul valid
int find_nonce(BYTE *difficulty, uint32_t max_nonce, BYTE *block_content, size_t current_length, BYTE *block_hash, uint32_t *valid_nonce) {
    BYTE *device_diffculty, *device_block_content, *device_found_hash;
    int *device_found_flag;
    uint32_t *device_valid_nonce;
    int found_flag = 0;

    // Alocam memorie pe GPU
    hipMalloc(&device_diffculty, SHA256_HASH_SIZE);
    hipMalloc(&device_block_content, BLOCK_SIZE);
    hipMalloc(&device_found_flag, sizeof(int));
    hipMalloc(&device_valid_nonce, sizeof(uint32_t));
    hipMalloc(&device_found_hash, SHA256_HASH_SIZE);

    // Copiem datele de pe gazda pe GPU
    hipMemcpy(device_diffculty, difficulty, SHA256_HASH_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(device_block_content, block_content, current_length, hipMemcpyHostToDevice);
    hipMemcpy(device_found_flag, &found_flag, sizeof(int), hipMemcpyHostToDevice);

    // Calculeaza cate blocuri si threaduri lansam
    size_t threads_per_block = 256;
    size_t launch_blocks = (static_cast<size_t>(max_nonce) + threads_per_block) / threads_per_block;

    // Calculam nonce-ul valid
    find_valid_nonce<<<launch_blocks, threads_per_block>>>(device_diffculty, device_block_content, current_length, max_nonce, device_found_flag, (uint32_t*)device_valid_nonce, device_found_hash);
    hipDeviceSynchronize();

    // Copiem rezultatele inapoi pe gazda
    hipMemcpy(&found_flag, device_found_flag, sizeof(int), hipMemcpyDeviceToHost);

    // Daca am gasit un nonce valid, copiem nonce-ul si hash-ul gasit inapoi pe gazda
    if (found_flag) {
        hipMemcpy(valid_nonce, device_valid_nonce, sizeof(uint32_t), hipMemcpyDeviceToHost);
        hipMemcpy(block_hash, device_found_hash, SHA256_HASH_SIZE, hipMemcpyDeviceToHost);
    }

    // Eliberam memoria GPU
    hipFree(device_diffculty);
    hipFree(device_block_content);
    hipFree(device_found_flag);
    hipFree(device_valid_nonce);
    hipFree(device_found_hash);

    // Returnam 0 daca am gasit un nonce valid, altfel 1
    if (found_flag) {
        return 0;
    } else {
        return 1;
    }
}

__global__ void dummy_kernel() {}

// Warm-up function
void warm_up_gpu() {
    BYTE *dummy_data;
    hipMalloc((void **)&dummy_data, 256);
    dummy_kernel<<<1, 1>>>();
    hipDeviceSynchronize();
    hipFree(dummy_data);
}
